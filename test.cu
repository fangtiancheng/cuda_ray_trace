#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "mydef.h"
__global__
void kernel(float* dev_rand){
    hiprandStateXORWOW_t rand_state;
    u64 x = threadIdx.x + blockIdx.x * blockDim.x;
    u64 y = threadIdx.y + blockIdx.y * blockDim.y;
    u64 offset = x + y * blockDim.x * gridDim.x;
    u64 seed = offset;
    hiprand_init(seed, 0, 0, &rand_state);
    dev_rand[offset] = hiprand_uniform(&rand_state);
}
int main(){
    const int image_width = 16 * 16;
    const int image_height = 16 * 16;
    dim3 grids(image_width / 16, image_height / 16);
    dim3 threads(16,16);
    float* dev_rand;
    HANDLE_ERROR( hipMalloc( (void**)&dev_rand, image_width*image_height*sizeof(float)));
    kernel<<<grids,threads>>>(dev_rand);
    float* host_rand;
    host_rand = (float*) malloc(image_width*image_height*sizeof(float));
    HANDLE_ERROR( hipMemcpy(host_rand, dev_rand, image_width*image_height*sizeof(float), hipMemcpyDeviceToHost) );
    hipFree(dev_rand);
    for(size_t i = 0;i < image_width*image_height&&i<100;i++){
        printf("%f\n", host_rand[i]);
    }
    free(host_rand);
    return 0;
}